#include "hip/hip_runtime.h"
#include"convay_game.h"

#include<hip/hip_runtime.h>
#include<>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include<stdlib.h>
#include<time.h>


#define THREADS_PER_BLOCK 10

struct neighbours {
	int leftIdx;
	int topIdx;
	int topLeftIdx;
	int topRightIdx;
	int rightIdx;
	int bottomIdx;
	int bottomLeftIdx;
	int bottomRightIdx;
};

__device__
int getCount(int const* board, int tid,int width,long size,neighbours n) {

	int cnt{ 0 };

	// check the corners and boundaries:
	if (tid == 0) {
		cnt += board[n.rightIdx] + board[n.bottomIdx] + board[n.bottomRightIdx];
		return cnt;
	}

	// bottom left corner:
	if (tid == (size - width)) {
		cnt += board[n.topIdx] + board[n.topRightIdx] + board[n.rightIdx];
		return cnt;
	}
	//	top right corner:
	if (tid == (width - 1)) {
		cnt += board[n.leftIdx] + board[n.bottomLeftIdx] + board[n.bottomIdx];
		return cnt;
	}
	//	bottom right corner:
	if (tid == (size - 1)) {
		cnt += board[n.topIdx] + board[n.topLeftIdx] + board[n.leftIdx];
		return cnt;
	}
	// left boundary of the board:
	if ((tid % width) == 0) {
		cnt += board[n.topIdx] + board[n.topRightIdx]
			+ board[n.rightIdx] + board[n.bottomRightIdx] + board[n.bottomIdx];
		return cnt;
	}
	// top boundary of the board:
	if ((tid >= 0) && (tid <= (width - 1))) {
		cnt += board[n.leftIdx] + board[n.bottomLeftIdx]
			+ board[n.bottomIdx] + board[n.bottomRightIdx] + board[n.rightIdx];
		return cnt;
	}
	// right boundary of the board:
	if (((tid + 1) % width) == 0) {
		cnt += board[n.topIdx] + board[n.topLeftIdx]
			+ board[n.leftIdx] + board[n.bottomLeftIdx] + board[n.bottomIdx];
		return cnt;
	}
	// bottom boundary of the board:
	if ((tid >= (size - width)) && (tid <= (size - 1))) {
		cnt += board[n.leftIdx] + board[n.topLeftIdx]
			+ board[n.topIdx] + board[n.topRightIdx] + board[n.rightIdx];
		return cnt;
	}

	cnt += board[n.leftIdx] + board[n.topLeftIdx] + board[n.topIdx] + board[n.topRightIdx]
		+ board[n.rightIdx] + board[n.bottomRightIdx] + board[n.bottomIdx] + board[n.bottomLeftIdx];
	return cnt;


}

__global__
void convay_kernel(int* outBoard, int const* inBoard, long long size) {
	unsigned int const tidx = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int const tidy = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int const tid = tidx + tidy * gridDim.x * blockDim.x;

	if (tid >= size)return;

	// width of the board:
	int const width = gridDim.x * blockDim.x;

	// compute indices of all neighbors:
	neighbours n;
	n.leftIdx = tid - 1;
	n.topIdx = tid - width;
	n.topLeftIdx = n.topIdx - 1;
	n.topRightIdx = n.topIdx + 1;
	n.rightIdx = tid + 1;
	n.bottomIdx = tid + width;
	n.bottomLeftIdx = n.bottomIdx - 1;
	n.bottomRightIdx = n.bottomIdx + 1;

	int cnt = getCount(inBoard, tid, width, size,n);

	if (inBoard[tid] == 0)
		outBoard[tid] = (int)(cnt == 3);
	else
		outBoard[tid] = (int)((cnt == 2) || (cnt == 3));

}


__global__
void init_random(unsigned int seed, hiprandState_t* states,long size) {
	unsigned int const tidx = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int const tidy = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int const tid = tidx + tidy * gridDim.x * blockDim.x;
	if (tid >= size)return;

	hiprand_init(seed, tid, 0, &states[tid]);
}

__global__
void generateBoard(int* board, hiprandState_t* states, long size) {
	unsigned int const tidx = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int const tidy = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int const tid = tidx + tidy * gridDim.x * blockDim.x;
	
	if (tid >= size)return;
	
	board[tid] = hiprand(&states[tid])%2;

}


void ConvayGame::print(int* board, long counter) {
	std::size_t const totalSize = boardSize_ * boardSize_;
	std::cout << counter << ".generation\n| ";
	for (std::size_t t = 0; t < totalSize; ++t) {
		if ((t > 0) && (t % boardSize_) == 0)
			std::cout << "|\n| ";
		std::cout << board[t] << " ";
	}
	std::cout << "|\n";
}


void ConvayGame::initBoard(int* DBoard) {
	dim3 const blockSize(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
	dim3 const gridSize = dim3((boardSize_ + blockSize.x - 1) / blockSize.x,
		(boardSize_ + blockSize.y - 1) / blockSize.y);
	long const totalSize = boardSize_ * boardSize_;

	hiprandState_t* states;
	hipMalloc((void**)&states, sizeof(hiprandState_t) * totalSize);
	init_random<<<gridSize,blockSize>>>(time(0), states, totalSize);
	generateBoard<<<gridSize,blockSize>>>(DBoard, states, totalSize);
	hipFree(states);
}

void ConvayGame::singleGeneration(int* outDBoard, int const* inDBoard) {
	dim3 const blockSize(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
	dim3 const gridSize = dim3((boardSize_ + blockSize.x - 1) / blockSize.x,
		(boardSize_ + blockSize.y - 1) / blockSize.y);
	long const totalSize = boardSize_ * boardSize_;
	convay_kernel<<<gridSize,blockSize>>>(outDBoard, inDBoard, totalSize);
}



void ConvayGame::play() {
	long const totalSize = boardSize_ * boardSize_;

	int* hBoard = (int*)malloc(sizeof(int) * totalSize);
	int* dBoard;
	int* dROBoard;
	hipMalloc((void**)&dBoard, sizeof(int) * totalSize);
	hipMalloc((void**)&dROBoard, sizeof(int) * totalSize);

	initBoard(dROBoard);
	hipMemcpy(hBoard, dROBoard, sizeof(int) * totalSize,
		hipMemcpyKind::hipMemcpyDeviceToHost);
	print(hBoard, 0);
	for (std::size_t t = 1; t < generations_; ++t) {
		singleGeneration(dBoard, dROBoard);
		hipMemcpy(hBoard, dBoard,sizeof(int)*totalSize,
			hipMemcpyKind::hipMemcpyDeviceToHost);
		print(hBoard, t);
		swap(dBoard,dROBoard);
	}


	free(hBoard);
	hipFree(dBoard);
	hipFree(dROBoard);

}